#include "hip/hip_runtime.h"
#include "particle_box.h"

void particle_box_init_host(particle_box_t &p, size_t capacity) {
  p.particles = new particle_t[capacity];
  p.capacity = capacity;
}

__device__ void particle_box_init_device(particle_box_t p, size_t capacity) {
  p.capacity = capacity;
  hipMalloc(&p.particles, sizeof(particle_t) * capacity);
}

void particle_box_realloc_host(particle_box_t &p, size_t capacity) {
  if (capacity <= p.capacity) {
    return;
  }
  particle_t *new_particles = new particle_t[capacity];
  p.capacity = capacity;
  for (size_t idx = 0; idx < p.particle_count; idx++) {
    new_particles[idx] = p.particles[idx];
  }
  delete[] p.particles;
  p.particles = new_particles;
}

__device__ void particle_box_realloc_device(particle_box_t p, size_t capacity) {
  if (capacity <= p.capacity) {
    return;
  }
  particle_t *new_particles;
  hipMalloc(&new_particles, sizeof(particle_t) * capacity);
  for (size_t idx = 0; idx < p.particle_count; idx++) {
    new_particles[idx] = p.particles[idx];
  }
  hipFree(p.particles);
  p.particles = new_particles;
}

void particle_box_add_particle_host(particle_box_t &box, double radius) {
  if (box.capacity >= box.particle_count) {
    particle_box_realloc_host(box, box.particle_count * 2);
  }

  bool intersects = true;
  particle_t *p = box.particles + box.particle_count;
  p->radius = radius;
  p->valid = true;
  particle_init_host(box.particles[box.particle_count]);
  do {
    intersects = false;
    random_particle_pos(*p, box.dimensions);

    for (size_t i = 0; i < box.particle_count; i++) {
      if (particle_intersects(box.particles[i], *p)) {
        intersects = true;
        break;
      }
    }
  } while (intersects);
  box.particle_count++;
}

__global__ void assign_patch(particle_t *p, patch_t *patches) {
  p->patches = patches;
}

particle_box_t make_box(particle_box_t const &box) {
  particle_box_t res{
      .dimensions = box.dimensions,
      .particle_count = box.particle_count,
      .capacity = box.capacity,
  };

  hipMalloc(&res.particles, sizeof(particle_t) * box.capacity);
  hipMemcpy(res.particles, box.particles,
             sizeof(particle_t) * box.particle_count, hipMemcpyHostToDevice);
  for (size_t i = 0; i < box.particle_count; i++) {
    patch_t *tmp;
    hipMalloc(&tmp, sizeof(patch_t) * box.particles[i].patch_count);
    hipMemcpy(tmp, box.particles[i].patches,
               sizeof(patch_t) * box.particles[i].patch_count,
               hipMemcpyHostToDevice);
    assign_patch<<<1, 1>>>(res.particles + i, tmp);
  }

  return res;
}

__global__ void freePatches(particle_t *p) { hipFree(p->patches); }

__host__ __device__ void particle_box_free_particles_device(particle_box_t p) {
  for (size_t idx = 0; idx < p.particle_count; idx++) {
    freePatches<<<1, 1>>>(p.particles + idx);
  }
  hipFree(p.particles);
  p.particle_count = 0;
  p.capacity = 0;
}
