#include "hip/hip_runtime.h"
#include "particle.h"
#include "vec.h"

#include <random>

void particle_t::random_particle_pos(double3 dimensions) {
  std::random_device r;
  std::mt19937 re(r());

  std::uniform_real_distribution<double> unif_x(0, dimensions.x);
  std::uniform_real_distribution<double> unif_y(0, dimensions.y);
  std::uniform_real_distribution<double> unif_z(0, dimensions.z);

  pos.x = unif_x(re);
  pos.y = unif_y(re);
  pos.z = unif_z(re);
};

double4 particle_t::random_particle_orient(double const angle, int axis) {
  double4 rotation{
      .x = cos(angle / 2),
      .y = 0,
      .z = 0,
      .w = 0,
  };

  switch (axis) {
  case 2:
    rotation.w = sin(angle / 2);
    break;
  case 1:
    rotation.z = sin(angle / 2);
    break;
  default:
    rotation.x = sin(angle / 2);
    break;
  };

  return rotation;
};

void particle_t::rotate(double4 const rot) {
  double4 conj = {
      .x = rot.x,
      .y = -rot.y,
      .z = -rot.z,
      .w = -rot.w,
  };

  orient = rot * (orient * conj);
  for (size_t i = 0; i < patch_count; i++) {
    patches[i].pos = rot * (patches[i].pos * conj);
  }
}

void particle_t::random_particle_pos(rng_gen &rng_x, rng_gen &rng_y,
                                     rng_gen &rng_z, std::mt19937 &re) {
  pos.x = rng_x(re);
  pos.y = rng_y(re);
  pos.z = rng_z(re);
}

__host__ __device__ double particle_t::interact(particle_t const &rhs,
                                                double const cosmax,
                                                double const epsilon) {
  double3 dist = normalize((double3){
      .x = rhs.pos.x - pos.x,
      .y = rhs.pos.y - pos.y,
      .z = rhs.pos.z - pos.z,
  });
  double result = 0;

  for (size_t i = 0; i < patch_count; i++) {
    patch_t const &p = patches[i];
    double3 p_pos = {p.pos.y, p.pos.z, p.pos.w};
    double p_cos = dot(p_pos, dist);

    if (p_cos < cosmax) {
      continue;
    }

    for (size_t j = 0; j < rhs.patch_count; j++) {
      patch_t const &q = rhs.patches[j];
      double3 q_pos = {q.pos.y, q.pos.z, q.pos.w};
      double q_cos = -dot(q_pos, dist);
      if (q_cos < cosmax) {
        continue;
      }

      result += epsilon;
    }
  }

  return result;
}
