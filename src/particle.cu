#include "hip/hip_runtime.h"
#include "particle.h"
#include "vec.h"

#include <random>

void particle_t::random_particle_pos(double3 dimensions) {
  std::random_device r;
  std::mt19937 re(r());

  std::uniform_real_distribution<double> unif_x(0, dimensions.x);
  std::uniform_real_distribution<double> unif_y(0, dimensions.y);
  std::uniform_real_distribution<double> unif_z(0, dimensions.z);

  pos.x = unif_x(re);
  pos.y = unif_y(re);
  pos.z = unif_z(re);
};

double4 particle_t::random_particle_orient(double const angle, int axis) {
  double4 rotation{
      .x = cos(angle / 2),
      .y = 0,
      .z = 0,
      .w = 0,
  };

  switch (axis) {
  case 2:
    rotation.w = sin(angle / 2);
    break;
  case 1:
    rotation.z = sin(angle / 2);
    break;
  default:
    rotation.x = sin(angle / 2);
    break;
  };

  return rotation;
};

void particle_t::rotate(double4 const rot) {
  double4 conj = {
      .x = rot.x,
      .y = -rot.y,
      .z = -rot.z,
      .w = -rot.w,
  };

  orient = rot * (orient * conj);
  for (size_t i = 0; i < patch_count; i++) {
    patches[i].pos = rot * (patches[i].pos * conj);
  }
}

void particle_t::random_particle_pos(rng_gen &rng_x, rng_gen &rng_y,
                                     rng_gen &rng_z, std::mt19937 &re) {
  pos.x = rng_x(re);
  pos.y = rng_y(re);
  pos.z = rng_z(re);
}

__host__ __device__ double particle_t::interact(particle_t const &rhs,
                                                double const cosmax,
                                                double const epsilon) const {
  double3 dist = normalize((double3){
      .x = pos.x - rhs.pos.x,
      .y = pos.x - rhs.pos.y,
      .z = pos.x - rhs.pos.z,
  });

  for (size_t i = 0; i < patch_count; i++) {
    patch_t const &p = patches[i];
    double3 p_pos = normalize((double3){
        p.pos.y,
        p.pos.z,
        p.pos.w,
    });
    double p_cos = dot(p_pos, dist);

    if (p_cos < cosmax) {
      continue;
    }

    for (size_t j = 0; j < rhs.patch_count; j++) {
      patch_t const &q = rhs.patches[j];
      double3 q_pos = normalize((double3){
          q.pos.y,
          q.pos.z,
          q.pos.w,
      });
      double q_cos = -dot(q_pos, dist);
      if (q_cos < cosmax) {
        continue;
      }

      return epsilon;
    }
  }

  return 0;
}

void particle_t::add_patch(patch_t const &p) {
  if (patch_count >= DEFAULT_CAPACITY) {
    return;
  }
  patches[patch_count] = p;
  patch_count++;
}
