#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <map>
#include <stdint.h>

#include <stdio.h>

void ShannonEntropy(int *data, int N, int &min, int &max, float &entropy);

__global__ void setup_kernel(hiprandState *state, uint64_t seed) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, tid, 0, &state[tid]);
}

__global__ void generate_randoms(hiprandState *globalState, float *randoms) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hiprandState localState = globalState[tid];
  randoms[tid * 2 + 0] = hiprand_uniform(&localState);
  randoms[tid * 2 + 1] = hiprand_uniform(&localState);
}

int main() {
  printf("\nTwoStepRandom\n");
  int threads = 256;
  int blocks = 5120;
  int threadCount = blocks * threads;
  int N = blocks * threads * 2;

  hiprandState *dev_curand_states;
  float *randomValues;
  float *host_randomValues;
  int *host_int;

  float time_elapsed_setup;
  float time_elapsed;
  hipEvent_t startTime;
  hipEvent_t stopTime;
  hipStream_t computeStream;

  // Init host memory
  host_randomValues = (float *)malloc(N * sizeof(float));
  host_int = (int *)malloc(N * sizeof(float));

  // Init device memory
  hipMalloc(&dev_curand_states, threadCount * sizeof(hiprandState));
  hipMalloc(&randomValues, N * sizeof(float));

  hipEventCreate(&startTime);
  hipEventCreate(&stopTime);
  hipStreamCreateWithFlags(&computeStream, hipStreamNonBlocking);

  //  ----- Setup seeds -----
  hipEventRecord(startTime, computeStream);

  setup_kernel<<<blocks, threads, 0, computeStream>>>(dev_curand_states,
                                                      time(NULL));

  hipEventRecord(stopTime, computeStream);
  hipEventSynchronize(stopTime);
  hipEventElapsedTime(&time_elapsed_setup, startTime, stopTime);

  // ----- Generate random numbers -----
  hipEventRecord(startTime, computeStream);

  // Needs both read and write from global memory
  generate_randoms<<<blocks, threads, 0, computeStream>>>(dev_curand_states,
                                                          randomValues);

  hipEventRecord(stopTime, computeStream);
  hipEventSynchronize(stopTime);

  hipEventElapsedTime(&time_elapsed, startTime, stopTime);

  // ----- Concluding Steps -----

  hipMemcpy(host_randomValues, randomValues, N * sizeof(float),
             hipMemcpyDeviceToHost);

  // Convert floats to ints for the shannnon entropy function
  for (int i = 0; i < N; ++i) {
    // Print a few values out
    if (i < 8) {
      printf("%.3f, ", host_randomValues[i]);
    }

    host_int[i] = (int)(host_randomValues[i] * 10000.0f);
  }

  printf("\n");
  printf("Elapsed time setup    %9.3f\n", time_elapsed_setup);
  printf("Elapsed time generate %9.3f\n", time_elapsed);

  int min, max;
  float entropy;
  ShannonEntropy(host_int, N, min, max, entropy);
  printf("Shannon Entropy <%6.3f>\n", entropy);

  hipFree(dev_curand_states);
  hipFree(randomValues);
  free(host_randomValues);
  free(host_int);

  return 0;
}

void ShannonEntropy(int *data, int N, int &min, int &max, float &entropy) {
  entropy = 0; // Init
  min = UINT_MAX;
  max = 0;

  std::map<int, long> counts;
  typename std::map<int, long>::iterator it;

  for (int dataIndex = 0; dataIndex < N; dataIndex++) {
    int dValue = data[dataIndex];
    if (dValue < min) {
      min = dValue;
    }
    if (dValue > max) {
      max = dValue;
    }
    counts[dValue]++;
  }

  it = counts.begin();
  while (it != counts.end()) {
    float p_x = (float)it->second / N;
    if (p_x > 0)
      entropy -= (float)(p_x * log(p_x) / log(2));
    it++;
  }
}
