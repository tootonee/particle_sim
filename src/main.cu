#include "hip/hip_runtime.h"
#include "particle.h"
#include "particle_box.h"
#include <stdio.h>

__global__ void print_particle(particle_box_t box) {
  int id = threadIdx.x;
  double3 const v = box.particles[id].pos;
  printf("(%lf, %lf, %lf)\n", v.x, v.y, v.z);
  // hipFree(box.particles[id].patches);
}

int main() {
  particle_box_t box{};
  box.dimensions = {10, 50, 10};
  particle_box_init_host(box, 512);

  // particle_t p1{.radius = 1.0F};
  // particle_t p2{.radius = 1.0F};

  for (size_t i = 0; i < 512; i++) {
    // std::cout << "I = " << i << std::endl;
    particle_box_add_particle_host(box, 1);
  }

  // for (size_t i = 0; i < 100; i++) {
  //   particle_box_add_particle_host(box, 1);
  // }

  // for (size_t i = 0; i < box.particle_count; i++) {
  //   double3 const v = b.particles[i].pos;
  //   std::cout << '(' << v.x << ", " << v.y << ", " << v.z << ")," <<
  //   std::endl;
  // }

  particle_box_t b = make_box(box);
  hipDeviceSynchronize();

  print_particle<<<1, 512>>>(b);

  hipDeviceSynchronize();

  particle_box_free_particles_host(box);
  particle_box_free_particles_device(b);

  hipDeviceSynchronize();

  return 0;
}
