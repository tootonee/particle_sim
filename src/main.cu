#include "hip/hip_runtime.h"
#include "particle.h"
#include "particle_box.h"
#include <iostream>
#include <random>
#include <stdio.h>

// __global__ void print_particle(particle_box_t box) {
//   int id = threadIdx.x;
//   double3 const v = box.particles[id].pos;
//   printf("(%lf, %lf, %lf)\n", v.x, v.y, v.z);
//   // hipFree(box.particles[id].patches);
// }

int main() {
  particle_box_t box{};
  box.dimensions = {11, 10, 10};
  particle_box_init_host(box, 512);

  // particle_t p1{.radius = 1.0F};
  // particle_t p2{.radius = 1.0F};

  std::random_device r;
  std::mt19937 re(r());

  std::uniform_real_distribution<double> unif_x(0, box.dimensions.x);
  std::uniform_real_distribution<double> unif_y(0, box.dimensions.y);
  std::uniform_real_distribution<double> unif_z(0, box.dimensions.z);

  for (size_t i = 0; i < 1024; i++) {
    std::cout << "I = " << i << std::endl;
    particle_box_add_particle_host(box, 0.5, unif_x, unif_y, unif_z, re);
  }

  for (size_t i = 0; i < box.particle_count; i++) {
    double3 const v = box.particles[i].pos;
    std::cout << '(' << v.x << ", " << v.y << ", " << v.z << ")," << std::endl;
  }

  particle_box_free_particles_host(box);

  // particle_box_t b = make_box(box);
  // hipDeviceSynchronize();

  // print_particle<<<1, 512>>>(b);

  // hipDeviceSynchronize();

  // particle_box_free_particles_host(box);
  // particle_box_free_particles_device(b);

  // hipDeviceSynchronize();

  return 0;
}
