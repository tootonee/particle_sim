#include "hip/hip_runtime.h"
#include "cell_view.h"
#include "particle.h"
#include "particle_box.h"
#include "pdb_export.h"

#include <algorithm>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <map>
#include <ostream>
#include <random>
#include <sstream>
#include <vector>

constexpr size_t PARTICLE_COUNT = 500;
constexpr size_t MOVES_PER_ITER = 200;
constexpr size_t ITERATIONS = 10'000;
constexpr size_t ITERATIONS_PER_EXPORT = 10;
constexpr size_t ITERATIONS_PER_GRF_EXPORT = 1000;
constexpr double TEMPERATURE = 1.5;

std::map<double, double>
do_distr(cell_view_t const &view, double const rho = 0.5L,
         double const start = 1L, double const dr = 0.01L,
         double const max_r = 5L, double const samples = 50) {
  std::map<double, double> distr{};
  double radius = start;
  double v_old = 0;
  double v_new = radius * radius * radius;
  std::random_device r;
  std::mt19937 re(r());

  std::uniform_real_distribution<double> unif_x(0, 0.999);

  while (radius < max_r) {
    double num = 0.0F;
    for (size_t s_idx = 0; s_idx < samples; s_idx++) {
      size_t const p_idx = unif_x(re) * view.box.particle_count;
      num += view.particles_in_range(p_idx, radius, radius + dr);
    }
    v_old = v_new;
    radius += dr;
    v_new = radius * radius * radius;
    double const val = 3 * num / (4 * M_PI * rho * (v_new - v_old));
    distr[radius] = val / samples;
  }
  return distr;
}

int main() {
  std::random_device r;
  std::mt19937 re(r());

  std::uniform_real_distribution<double> unif_x(0, 10);
  std::uniform_real_distribution<double> unif_y(0, 10);
  std::uniform_real_distribution<double> unif_z(0, 10);
  cell_view_t view({10, 10, 10}, 10);

  std::vector<std::pair<size_t, size_t>> intersects{};

  for (size_t i = 1; i <= PARTICLE_COUNT; i++) {
    view.add_particle_random_pos(0.5, unif_x, unif_y, unif_z, re);
  }

  std::uniform_real_distribution<double> unif_r(0, 1);

  double const rho =
      view.box.particle_count /
      (view.box.dimensions.x * view.box.dimensions.y * view.box.dimensions.z);
  std::map<double, double> distr{};
  double init_energy = view.total_energy(0.2, -1);

  for (size_t iters = 0; iters <= ITERATIONS; iters++) {
    if (iters % ITERATIONS_PER_GRF_EXPORT == 0) {
      std::map<double, double> tmp_distr =
          do_distr(view, rho, 1, 0.02L, 5, 150);
      for (const auto &[radius, value] : tmp_distr) {
        distr[radius] += value;
      }
    }

    if (iters % ITERATIONS_PER_EXPORT == 0) {
      const size_t idx = iters / ITERATIONS_PER_EXPORT;
      char buf[16];
      std::sprintf(buf, "data/%06li.pdb", idx);
      export_particles_to_pdb(view.box, buf);
      std::cout << "I = " << idx << ", energy = " << init_energy << std::endl;
    }

    for (size_t i = 0; i < MOVES_PER_ITER; i++) {
      size_t const p_idx =
          static_cast<size_t>(unif_r(re) * view.box.particle_count) %
          view.box.particle_count;
      double3 const old_pos = view.box.particles[p_idx].pos;
      particle_t &part = view.box.particles[p_idx];

      double3 const new_pos =
          view.try_random_particle_disp(p_idx, unif_r, re, 1);

      if (new_pos.x == -1) {
        continue;
      }
      //
      // double const old_energy = view.particle_energy_square_well(part, 0.2,
      // -1);
      // // double const old_energy =
      // // view.particle_energy_square_well_device(part, 1.5);
      //
      // part.pos = new_pos;
      // double new_energy = view.particle_energy_square_well(part, 0.2, -1);
      // // double const new_energy =
      // //     view.particle_energy_square_well_device(part, 1.5);
      // part.pos = old_pos;
      //
      // double prob = exp((old_energy - new_energy) / TEMPERATURE);
      // if (unif_r(re) <= prob && new_energy < old_energy) {
      //   continue;
      // }
      // init_energy += old_energy - new_energy;
      view.remove_particle(view.box.particles[p_idx]);
      part.pos = new_pos;
      view.box.update_particle(p_idx);
      view.add_particle(view.box.particles[p_idx]);
    }
  }

  std::ofstream other_file("output.dat");
  other_file << std::fixed << std::setprecision(6);
  double const coeff = ITERATIONS / ITERATIONS_PER_GRF_EXPORT + 1;
  for (const auto &[r, val] : distr) {
    double const real_val = val / coeff;
    if (real_val <= 0.1) {
      continue;
    }
    other_file << r << "    " << real_val << std::endl;
  }

  view.free();

  return 0;
}
