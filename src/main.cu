#include "hip/hip_runtime.h"
#include "cell_view.h"
#include "particle.h"
#include "particle_box.h"
#include "pdb_export.h"

#include <algorithm>
#include <fstream>
#include <iomanip>
#include <iostream>
#include <map>
#include <ostream>
#include <random>
#include <sstream>
#include <vector>

// __global__ void print_particle(particle_box_t box) {
//   int id = threadIdx.x;
//   double3 const v = box.particles[id].pos;
//   printf("(%lf, %lf, %lf)\n", v.x, v.y, v.z);
//   // hipFree(box.particles[id].patches);
// }

constexpr size_t PARTICLE_COUNT = 500;
/* constexpr size_t ITERATIONS = 10'000; */
constexpr size_t ITERATIONS = 10'000;
constexpr size_t ITERATIONS_PER_EXPORT = 100;
constexpr size_t ITERATIONS_PER_GRF_EXPORT = 100;
constexpr double TEMPERATURE = 1.5;
/* constexpr double TEMPERATURE = 2.15L; */
/* constexpr double BOLTZMANN_C = 1.380649e-23L; */
/* constexpr double BOLTZMANN_C = 1.380649e-1L; */
/* constexpr double BOLTZMANN_C = 1.0L; */

std::map<double, double> do_distr(cell_view_t const &view,
                                  double const rho = 0.5L,
                                  double const start = 1L,
                                  double const dr = 0.01L,
                                  double const max_r = 5L) {
  std::map<double, double> distr{};
  double radius = start - dr;
  double v_old = 0;
  double v_new = radius * radius * radius;

  while (radius < max_r) {
    double num = 0.0F;
    for (size_t p_idx = 0; p_idx < view.box.particle_count; p_idx++) {
      num += view.particles_in_range(p_idx, radius, radius + dr);
    }
    v_old = v_new;
    radius += dr;
    v_new = radius * radius * radius;
    double const val = 3 * num / (4 * M_PI * rho * (v_new - v_old));
    distr[radius] = val / view.box.particle_count;
  }
  return distr;
}

int main() {
  std::random_device r;
  std::mt19937 re(r());

  std::uniform_real_distribution<double> unif_x(0, 10);
  std::uniform_real_distribution<double> unif_y(0, 10);
  std::uniform_real_distribution<double> unif_z(0, 10);
  cell_view_t view({10, 10, 10}, 16);
  std::cout << "Box particles = " << view.box.particles << std::endl;

  std::vector<std::pair<size_t, size_t>> intersects{};

  for (size_t i = 0; i < PARTICLE_COUNT; i++) {
    std::cout << "I = " << i << std::endl;
    view.add_particle_random_pos(0.5, unif_x, unif_y, unif_z, re);
  }

  std::uniform_real_distribution<double> unif_r(0, 1);

  double const rho =
      view.box.particle_count /
      (view.box.dimensions.x * view.box.dimensions.y * view.box.dimensions.z);
  std::map<double, double> distr{};

  for (size_t iters = 0; iters <= ITERATIONS; iters++) {
    if (iters % ITERATIONS_PER_GRF_EXPORT == 0) {
      std::map<double, double> tmp_distr = do_distr(view, rho, 1, 0.05L);
      for (const auto &[radius, value] : tmp_distr) {
          distr[radius] += value;
      }
    }

    if (iters % ITERATIONS_PER_EXPORT == 0) {
      const size_t idx = iters / ITERATIONS_PER_EXPORT;
      char buf[16];
      std::sprintf(buf, "data/%06li.pdb", idx);
      export_particles_to_pdb(view.box, buf);

      std::cout << "I = " << idx << std::endl;
    }

    for (size_t i = 0; i < view.box.particle_count; i++) {
      size_t const p_idx = static_cast<size_t>(unif_r(re) * view.box.particle_count) % view.box.particle_count;
      double3 const old_pos = view.box.particles[p_idx].pos;
      particle_t &part = view.box.particles[p_idx];

      // double const old_energy = view.particle_energy_square_well(part, 0.2, 1);

      // double const old_energy =
      // view.particle_energy_square_well_device(part, 1.5);

      double3 const new_pos =
          view.try_random_particle_disp(p_idx, unif_r, re, 0.5);
      if (new_pos.x == -1) {
        continue;
      }
      // part.pos = new_pos;
      // double new_energy = view.particle_energy_square_well(part, 0.2, 1);
      // // double const new_energy =
      // //     view.particle_energy_square_well_device(part, 1.5);
      // part.pos = old_pos;

      // double prob = exp((old_energy - new_energy) / TEMPERATURE);
      // if (unif_r(re) >= prob) {
      //   continue;
      // }
      view.remove_particle(view.box.particles[p_idx]);
      part.pos = new_pos;
      view.box.update_particle(p_idx);
      view.add_particle(view.box.particles[p_idx]);
    }
  }

  std::ofstream other_file("output.dat");
  other_file << std::fixed << std::setprecision(6);
  double const coeff = (ITERATIONS / ITERATIONS_PER_GRF_EXPORT) + 1; 
  for (const auto &[r, val] : distr) {
    double const real_val = val / coeff;
    if (real_val <= 0.1) {
      continue;
    }
    other_file << r << "    " << real_val << std::endl;
  }

  // std::map<double, double> tmp_distr = do_distr(view, rho, 0.95, 0.01L);
  // for (const auto &[radius, value] : tmp_distr) {
  //     distr[radius] += value;
  // }
  // std::ofstream other_file("output.dat");
  // other_file << std::fixed << std::setprecision(6);
  // double const coeff = (ITERATIONS / ITERATIONS_PER_GRF_EXPORT) + 1; 
  // for (const auto &[r, val] : distr) {
  //   if (val <= 0.1) {
  //     continue;
  //   }
  //   other_file << r << "    " << val << std::endl;
  // }

  view.free();

  return 0;
}
